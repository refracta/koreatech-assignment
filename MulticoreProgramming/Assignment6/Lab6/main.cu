#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <omp.h>
#include <random>
#include <sstream>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"
#include "DS_timer.h"

#define USE_HOST
#define USE_OMP
#define TO_INDEX(row, col, width) ((row) * (width) + (col))
#define GET(pointer, row, col, width) (pointer[TO_INDEX(row, col, width)])
#define GET_B(pointer, row, col) GET(pointer, row, col, blockSize)

int row;
int k;
int col;
int blockSize;
int sizeA;
int sizeB;
int sizeC;
float *a, *b, *c, *h_c, *o_c;
float *d_a, *d_b;
float *d_c;
// __constant__ float d_a[32 * 32], d_b[32 * 32];

__global__ void matrixMul(const float *a, const float *b, float *c, int row, int k, int col) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;
    if (!(rowIx < row && colIx < col)) {
        return;
    }

    float sum = 0.0f;
    for (int kIx = 0; kIx < k; kIx++) {
        sum += __fmul_rn(a[rowIx * k + kIx], b[kIx * col + colIx]);
    }
    c[rowIx * col + colIx] = sum;
}

__global__ void matrixMulRR(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    unsigned int blockLimit = ceil(k / (float) blockSize);

    float result = 0;
    for (int block = 0; block < blockLimit; block++) {
        int offset = block * blockSize;

        GET_B(sharedA, threadRow, threadCol) = (rowIx < row && offset + threadCol < k) ?
                                               a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        GET_B(sharedB, threadRow, threadCol) = (colIx < col && offset + threadRow < k) ?
                                               b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, threadRow, i), GET_B(sharedB, i, threadCol));
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}

__global__ void matrixMulCC(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    unsigned int blockLimit = ceil(k / (float) blockSize);

    float result = 0;
    for (int block = 0; block < blockLimit; block++) {
        int offset = block * blockSize;

        GET_B(sharedA, threadCol, threadRow) = (rowIx < row && offset + threadCol < k) ?
                                               a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        GET_B(sharedB, threadCol, threadRow) = (colIx < col && offset + threadRow < k) ?
                                               b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, i, threadRow), GET_B(sharedB, threadCol, i));
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}

#define BLOCK_SIZE 16

__global__ void
matrixMulCC_V_STATIC_BLOCK(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    unsigned int blockLimit = ceil(k / (float) blockSize);

    float result = 0;
    for (int block = 0; block < blockLimit; block++) {
        int offset = block * blockSize;

        sharedA[threadCol][threadRow] = (rowIx < row && offset + threadCol < k) ?
                                        a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        sharedB[threadCol][threadRow] = (colIx < col && offset + threadRow < k) ?
                                        b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(sharedA[i][threadRow], sharedB[threadCol][i]);
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}

__global__ void
matrixMulCC_V_IDX_CHACHING(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;
    const unsigned int targetIdx = TO_INDEX(threadRow, threadCol, blockSize);
    const unsigned int limit = ceil(k / (float) blockSize) * blockSize;

    float result = 0;
    for (int offset = 0; offset < limit; offset += blockSize) {
        sharedA[targetIdx] = (rowIx < row && offset + threadCol < k) ?
                             a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        sharedB[targetIdx] = (colIx < col && offset + threadRow < k) ?
                             b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, i, threadRow), GET_B(sharedB, threadCol, i));
        }
        __syncthreads();
    }

    if (rowIx >= row || colIx >= col) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}


__global__ void matrixMulCC_V_FINAL(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    const unsigned int limit = ceil(k / (float) blockSize) * blockSize;

    float result = 0;
    for (int offset = 0; offset < limit; offset += blockSize) {
        GET_B(sharedA, threadCol, threadRow) = (rowIx < row && offset + threadCol < k) ?
                                               a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        GET_B(sharedB, threadCol, threadRow) = (colIx < col && offset + threadRow < k) ?
                                               b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, i, threadRow), GET_B(sharedB, threadCol, i));
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}


__global__ void matrixMulRC(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    unsigned int blockLimit = ceil(k / (float) blockSize);

    float result = 0;
    for (int block = 0; block < blockLimit; block++) {
        int offset = block * blockSize;

        GET_B(sharedA, threadRow, threadCol) = (rowIx < row && offset + threadCol < k) ?
                                               a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        GET_B(sharedB, threadCol, threadRow) = (colIx < col && offset + threadRow < k) ?
                                               b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, threadRow, i), GET_B(sharedB, threadCol, i));
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}

__global__ void matrixMulCR(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    unsigned int blockLimit = ceil(k / (float) blockSize);

    float result = 0;
    for (int block = 0; block < blockLimit; block++) {
        int offset = block * blockSize;

        GET_B(sharedA, threadCol, threadRow) = (rowIx < row && offset + threadCol < k) ?
                                               a[TO_INDEX(rowIx, offset + threadCol, k)] : 0;
        GET_B(sharedB, threadRow, threadCol) = (colIx < col && offset + threadRow < k) ?
                                               b[TO_INDEX(offset + threadRow, colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, i, threadRow), GET_B(sharedB, i, threadCol));
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}

DS_timer timer(TIMER_LIST_SIZE);

inline void initTimer() {
    for (int i = 0; i < TIMER_LIST_SIZE; i++) {
        timer.setTimerName(i, TIMER_NAME_LIST[i]);
    }
    timer.initTimers();
}


inline void initData(int argc, char **argv) {
    if (argc < 1 + 4) {
        printf("Usage: Execution_file [row] [k] [col] [blockSize]\n");
        exit(1);
    }
    row = atoi(argv[1]);
    k = atoi(argv[2]);
    col = atoi(argv[3]);
    blockSize = atoi(argv[4]);

    printf("[row, k, col, blockSize] = [%d, %d, %d, %d]\n", row, k, col, blockSize);

    a = new float[sizeA = row * k];
    b = new float[sizeB = k * col];
    c = new float[sizeC = row * col];
    h_c = new float[sizeC];
    o_c = new float[sizeC];

    std::random_device rd;
    std::mt19937 engine(rd());
    std::uniform_real_distribution<float> distribution(0.0f, 1.0f);

    for (int i = 0; i < sizeA; i++) {
        a[i] = distribution(engine);
    }
    for (int i = 0; i < sizeB; i++) {
        b[i] = distribution(engine);
    }
    std::fill(c, c + sizeC, 0.0f);
}

bool checkResult(const float *base, const float *target) {
    bool isValidData = true;
    for (int i = 0; i < sizeC; i++) {
        if (base[i] != target[i]) {
            printf("\t[%d] The result is not matched! (base, target)=(%f, %f)\n", i, base[i], target[i]);
            isValidData = false;
        }
    }

    if (isValidData) {
        printf("\tTarget data valid!\n");
    }

    return isValidData;
}

void checkGPUResult(const char *name, bool clearDeviceMemory = true) {
#ifdef USE_OMP
    printf("checkGPUResult(%s)\n", name);
    hipMemcpy(c, d_c, sizeC * sizeof(float), hipMemcpyDeviceToHost);
    checkResult(o_c, c);
    if (clearDeviceMemory) {
        hipMemcpy(d_c, c, sizeC * sizeof(float), hipMemcpyHostToDevice);
    }
#else
    printf("checkGPUResult(%s): Passed\n", name);
#endif
}

inline void freeData() {
    delete[] a;
    delete[] b;
    delete[] c;
    delete[] h_c;
    delete[] o_c;
}

std::string toReportFileName(int row, int k, int col, int blockSize) {
    std::ostringstream stream;
    stream << "report_" << row << "_" << k << "_" << col << "_" << blockSize << ".txt";
    return stream.str();
}

int main(int argc, char **argv) {
    initData(argc, argv);
    initTimer();

#ifdef USE_HOST
    timer.onTimer(TIMER_HOST);
    for (int rowIx = 0; rowIx < row; rowIx++) {
        for (int colIx = 0; colIx < col; colIx++) {
            float sum = 0.0f;
            for (int kIx = 0; kIx < k; kIx++) {
                sum += a[rowIx * k + kIx] * b[kIx * col + colIx];
            }
            h_c[rowIx * col + colIx] = sum;
        }
    }
    timer.offTimer(TIMER_HOST);
#endif

#ifdef USE_OMP
    timer.onTimer(TIMER_OMP);
#pragma omp parallel for
    for (int rowIx = 0; rowIx < row; rowIx++) {
        for (int colIx = 0; colIx < col; colIx++) {
            float sum = 0.0f;
            for (int kIx = 0; kIx < k; kIx++) {
                sum += a[rowIx * k + kIx] * b[kIx * col + colIx];
            }
            o_c[rowIx * col + colIx] = sum;
        }
    }
    timer.offTimer(TIMER_OMP);
#endif

#ifdef USE_HOST
    printf("checkResult(%s)\n", TIMER_OMP_NAME);
    checkResult(h_c, o_c);
#endif

    hipMalloc(&d_a, sizeA * sizeof(float));
    hipMalloc(&d_b, sizeB * sizeof(float));
    hipMalloc(&d_c, sizeC * sizeof(float));

    timer.onTimer(TIMER_DATA_TRANSFER_H2D);
    /* hipMemcpyToSymbol(HIP_SYMBOL(d_a), a, sizeA * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_b), b, sizeB * sizeof(float)); */
    hipMemcpy(d_a, a, sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeC * sizeof(float), hipMemcpyHostToDevice);
    timer.offTimer(TIMER_DATA_TRANSFER_H2D);

    dim3 dimGrid((unsigned int) ceil(row / blockSize), (unsigned int) ceil(col / blockSize));
    dim3 dimBlock(blockSize, blockSize);

    timer.onTimer(TIMER_KERNEL_BASIC);
    matrixMul <<<dimGrid, dimBlock,
    2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL_BASIC);
    checkGPUResult(TIMER_KERNEL_BASIC_NAME);

    timer.onTimer(TIMER_KERNEL_RR);
    matrixMulRR <<<dimGrid, dimBlock,
    2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col, blockSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL_RR);
    checkGPUResult(TIMER_KERNEL_RR_NAME);

    timer.onTimer(TIMER_KERNEL_RC);
    matrixMulRC <<<dimGrid, dimBlock,
    2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col, blockSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL_RC);
    checkGPUResult(TIMER_KERNEL_RC_NAME);

    timer.onTimer(TIMER_KERNEL_CR);
    matrixMulCR <<<dimGrid, dimBlock,
    2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col, blockSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL_CR);
    checkGPUResult(TIMER_KERNEL_CR_NAME);

    timer.onTimer(TIMER_KERNEL_CC);
    matrixMulCC <<<dimGrid, dimBlock,
    2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col, blockSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL_CC);
    checkGPUResult(TIMER_KERNEL_CC_NAME);

    timer.onTimer(TIMER_KERNEL_CC_V_FINAL);
    matrixMulCC_V_FINAL<<<dimGrid, dimBlock,
    2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col, blockSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL_CC_V_FINAL);
    checkGPUResult(TIMER_KERNEL_CC_V_FINAL_NAME);

    timer.onTimer(TIMER_DATA_TRANSFER_D2H);
    hipMemcpy(c, d_c, sizeC * sizeof(float), hipMemcpyDeviceToHost);
    timer.offTimer(TIMER_DATA_TRANSFER_D2H);

    timer.printTimer();

    timer.printToFile((char *) toReportFileName(row, k, col, blockSize).c_str());

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    freeData();

    return 0;
}
