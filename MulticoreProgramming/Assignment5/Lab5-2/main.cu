#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <omp.h>
#include <random>
#include <sstream>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"
#include "DS_timer.h"

#define TO_INDEX(row, col, width) ((row) * (width) + (col))
#define GET(pointer, row, col, width) (pointer[TO_INDEX(row, col, width)])
#define GET_B(pointer, row, col) GET(pointer, row, col, blockSize)

__global__ void matrixMul(const float *a, const float *b, float *c, int row, int k, int col, int blockSize) {
    unsigned int rowIx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int colIx = blockDim.y * blockIdx.y + threadIdx.y;

    extern __shared__ float sharedTotal[];
    float *sharedA = sharedTotal;
    float *sharedB = sharedA + blockSize * blockSize;

    unsigned int threadRow = threadIdx.x;
    unsigned int threadCol = threadIdx.y;

    unsigned int blockLimit = ceil(k / (float) blockSize);

    float result = 0;
    for (int block = 0; block < blockLimit; block++) {
        int offset = block * blockSize;

        GET_B(sharedA, threadCol, threadRow) = (rowIx < row && offset + threadCol < k) ? a[TO_INDEX(rowIx,
                                                                                                    offset + threadCol,
                                                                                                    k)] : 0;
        GET_B(sharedB, threadCol, threadRow) = (colIx < col && offset + threadRow < k) ? b[TO_INDEX(offset + threadRow,
                                                                                                    colIx, col)] : 0;
        __syncthreads();

        for (int i = 0; i < blockSize; i++) {
            result += __fmul_rn(GET_B(sharedA, i, threadRow), GET_B(sharedB, threadCol, i));
        }
        __syncthreads();
    }

    if (!(rowIx < row && colIx < col)) {
        return;
    }

    c[TO_INDEX(rowIx, colIx, col)] = result;
}

DS_timer timer(TIMER_LIST_SIZE);

inline void initTimer() {
    timer.setTimerName(TIMER_TOTAL, TIMER_TOTAL_NAME);
    timer.setTimerName(TIMER_KERNEL, TIMER_KERNEL_NAME);
    timer.setTimerName(TIMER_DATA_TRANSFER_H2D, TIMER_DATA_TRANSFER_H2D_NAME);
    timer.setTimerName(TIMER_DATA_TRANSFER_D2H, TIMER_DATA_TRANSFER_D2H_NAME);
    timer.setTimerName(TIMER_HOST, TIMER_HOST_NAME);
    timer.setTimerName(TIMER_OMP, TIMER_OMP_NAME);
    timer.initTimers();
}

int row;
int k;
int col;
int blockSize;
int sizeA;
int sizeB;
int sizeC;
float *a, *b, *c, *h_c, *o_c;
float *d_a, *d_b, *d_c;

inline void initData(int argc, char **argv) {
    if (argc < 1 + 4) {
        printf("Usage: Execution_file [row] [k] [col] [blockSize]\n");
        exit(1);
    }
    row = atoi(argv[1]);
    k = atoi(argv[2]);
    col = atoi(argv[3]);
    blockSize = atoi(argv[4]);

    printf("[row, k, col, blockSize] = [%d, %d, %d, %d]\n", row, k, col, blockSize);

    a = new float[sizeA = row * k];
    b = new float[sizeB = k * col];
    c = new float[sizeC = row * col];
    h_c = new float[sizeC];
    o_c = new float[sizeC];

    std::random_device rd;
    std::mt19937 engine(rd());
    std::uniform_real_distribution<float> distribution(0.0f, 1.0f);

    for (int i = 0; i < sizeA; i++) {
        a[i] = distribution(engine);
    }
    for (int i = 0; i < sizeB; i++) {
        b[i] = distribution(engine);
    }
    std::fill(c, c + sizeC, 0.0f);
}

inline void freeData() {
    delete[] a;
    delete[] b;
    delete[] c;
    delete[] h_c;
    delete[] o_c;
}

std::string toReportFileName(int row, int k, int col, int blockSize) {
    std::ostringstream stream;
    stream << "report_" << row << "_" << k << "_" << col << "_" << blockSize << ".txt";
    return stream.str();
}

int main(int argc, char **argv) {
    initData(argc, argv);
    initTimer();

    hipMalloc(&d_a, sizeA * sizeof(float));
    hipMalloc(&d_b, sizeB * sizeof(float));
    hipMalloc(&d_c, sizeC * sizeof(float));

    timer.onTimer(TIMER_TOTAL);

    timer.onTimer(TIMER_DATA_TRANSFER_H2D);
    hipMemcpy(d_a, a, sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeC * sizeof(float), hipMemcpyHostToDevice);
    timer.offTimer(TIMER_DATA_TRANSFER_H2D);

    dim3 dimGrid((unsigned int) ceil(row / blockSize), (unsigned int) ceil(col / blockSize));
    dim3 dimBlock(blockSize, blockSize);

    timer.onTimer(TIMER_KERNEL);
    matrixMul <<<dimGrid, dimBlock, 2 * blockSize * blockSize * sizeof(float)>>>(d_a, d_b, d_c, row, k, col, blockSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL);

    timer.onTimer(TIMER_DATA_TRANSFER_D2H);
    hipMemcpy(c, d_c, sizeC * sizeof(float), hipMemcpyDeviceToHost);
    timer.offTimer(TIMER_DATA_TRANSFER_D2H);

    timer.offTimer(TIMER_TOTAL);

    timer.onTimer(TIMER_HOST);
    for (int rowIx = 0; rowIx < row; rowIx++) {
        for (int colIx = 0; colIx < col; colIx++) {
            float sum = 0.0f;
            for (int kIx = 0; kIx < k; kIx++) {
                sum += a[rowIx * k + kIx] * b[kIx * col + colIx];
            }
            h_c[rowIx * col + colIx] = sum;
        }
    }
    timer.offTimer(TIMER_HOST);

    timer.onTimer(TIMER_OMP);
    #pragma omp parallel for
    for (int rowIx = 0; rowIx < row; rowIx++) {
        for (int colIx = 0; colIx < col; colIx++) {
            float sum = 0.0f;
            for (int kIx = 0; kIx < k; kIx++) {
                sum += a[rowIx * k + kIx] * b[kIx * col + colIx];
            }
            o_c[rowIx * col + colIx] = sum;
        }
    }
    timer.offTimer(TIMER_OMP);

    timer.printTimer();

    bool isValidData = true;
    for (int i = 0; i < sizeC; i++) {
        if (!(h_c[i] == c[i] && o_c[i] == c[i])) {
            printf("[%d] The result is not matched! (Host, OpenMP, CUDA)=(%f, %f, %f)\n", i, h_c[i], o_c[i], c[i]);
            isValidData = false;
        }
    }

    if (isValidData) {
        printf("CUDA & OpenMP works well!\n");
    }

    timer.printToFile((char *) toReportFileName(row, k, col, blockSize).c_str());

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    freeData();

    return 0;
}
