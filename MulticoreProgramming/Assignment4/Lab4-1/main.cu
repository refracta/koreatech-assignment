#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"
#include "DS_timer.h"

#define DEFAULT_DATA_SIZE (1024 * 1024)

__global__ void vectorAdd(const int *a, const int *b, int *c, int size) {
    unsigned int tID = blockIdx.x * blockDim.x + threadIdx.x;
    if (tID < size) {
        c[tID] = a[tID] + b[tID];
    }
}

DS_timer timer(TIMER_LIST_SIZE);

inline void initTimer() {
    timer.setTimerName(TIMER_TOTAL, TIMER_TOTAL_NAME);
    timer.setTimerName(TIMER_KERNEL, TIMER_KERNEL_NAME);
    timer.setTimerName(TIMER_DATA_TRANSFER_H2D, TIMER_DATA_TRANSFER_H2D_NAME);
    timer.setTimerName(TIMER_DATA_TRANSFER_D2H, TIMER_DATA_TRANSFER_D2H_NAME);
    timer.setTimerName(TIMER_HOST, TIMER_HOST_NAME);
    timer.initTimers();
}

int dataSize;
int *a, *b, *c, *h_c;
int *d_a, *d_b, *d_c;
long long memorySize;

inline void initData(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: Execution_file [dataSize]\n");
        exit(1);
    }
    dataSize = atoi(argv[1]) * DEFAULT_DATA_SIZE;
    memorySize = sizeof(int) * dataSize;

    printf("%d elements, memorySize = %lld bytes\n", dataSize, memorySize);

    a = new int[dataSize];
    b = new int[dataSize];
    c = new int[dataSize];
    h_c = new int[dataSize];

    for (int i = 0; i < dataSize; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }
}
inline void freeData() {
    delete[] a;
    delete[] b;
    delete[] c;
}

int main(int argc, char **argv) {
    initData(argc, argv);
    initTimer();

    hipMalloc(&d_a, memorySize);
    hipMalloc(&d_b, memorySize);
    hipMalloc(&d_c, memorySize);

    timer.onTimer(TIMER_TOTAL);

    timer.onTimer(TIMER_DATA_TRANSFER_H2D);
    hipMemcpy(d_a, a, memorySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, memorySize, hipMemcpyHostToDevice);
    timer.offTimer(TIMER_DATA_TRANSFER_H2D);

    dim3 dimGrid(ceil(dataSize / 1024.0), 1, 1);
    dim3 dimBlock(1024, 1, 1);

    timer.onTimer(TIMER_KERNEL);
    vectorAdd <<<dimGrid, dimBlock>>>(d_a, d_b, d_c, dataSize);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL);

    timer.onTimer(TIMER_DATA_TRANSFER_D2H);
    hipMemcpy(c, d_c, memorySize, hipMemcpyDeviceToHost);
    timer.offTimer(TIMER_DATA_TRANSFER_D2H);

    timer.offTimer(TIMER_TOTAL);

    timer.onTimer(TIMER_HOST);
    for (int i = 0; i < dataSize; i++) {
        h_c[i] = a[i] + b[i];
    }
    timer.offTimer(TIMER_HOST);

    timer.printTimer();

    bool isValidData = true;
    for (int i = 0; i < dataSize; i++) {
        if (h_c[i] != c[i]) {
            printf("[%d] The result is not matched! (%d, %d)\n", i, h_c[i], c[i]);
            isValidData = false;
        }
    }

    if (isValidData) {
        printf("GPU works well!\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    freeData();

    return 0;
}
