#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <string>
#include <format>
#include <sstream>
#include "hip/hip_runtime.h"
#include ""
#include "timer.h"
#include "DS_timer.h"

#define LIMIT (13 * 2)
#define FLATTEN_MATRIX_SIZE (1 << LIMIT)
// FLATTEN_MATRIX_SIZE = 8192 * 8192
#define MEMORY_SIZE (FLATTEN_MATRIX_SIZE * sizeof(int))
#define TO_INDEX(x, y, z, w, dimX, dimY, dimZ) (x + dimX * (y + dimY * (z + dimZ * w)))

__global__ void vectorAdd(const int *a, const int *b, int *c, int size) {
    unsigned int tID = TO_INDEX(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x);
    if (tID < size) {
        c[tID] = a[tID] + b[tID];
    }
}

DS_timer timer(TIMER_LIST_SIZE);

inline void initTimer() {
    timer.setTimerName(TIMER_TOTAL, TIMER_TOTAL_NAME);
    timer.setTimerName(TIMER_KERNEL, TIMER_KERNEL_NAME);
    timer.setTimerName(TIMER_DATA_TRANSFER_H2D, TIMER_DATA_TRANSFER_H2D_NAME);
    timer.setTimerName(TIMER_DATA_TRANSFER_D2H, TIMER_DATA_TRANSFER_D2H_NAME);
    timer.setTimerName(TIMER_HOST, TIMER_HOST_NAME);
    timer.initTimers();
}

int *a, *b, *c, *h_c;
int *d_a, *d_b, *d_c;

inline void initData() {
    printf("%d elements, memorySize = %lld bytes\n", FLATTEN_MATRIX_SIZE, MEMORY_SIZE);

    a = new int[FLATTEN_MATRIX_SIZE];
    b = new int[FLATTEN_MATRIX_SIZE];
    c = new int[FLATTEN_MATRIX_SIZE];
    h_c = new int[FLATTEN_MATRIX_SIZE];

    for (int i = 0; i < FLATTEN_MATRIX_SIZE; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }
}

inline void freeData() {
    delete[] a;
    delete[] b;
    delete[] c;
}

std::string toReportFileName(int gridX, int gridY, int blockX, int blockY) {
    std::ostringstream stream;
    stream << "report_" << gridX << "_" << gridY << "_" << blockX << "_" << blockY << ".txt";
    return stream.str();
}

bool runVectorAdd(dim3 dimGrid, dim3 dimBlock) {
    printf("runVectorAdd(dimGrid={%d, %d, %d}, dimBlock={%d, %d, %d})\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x,
           dimBlock.y, dimBlock.z);

    timer.initTimers();
    hipMalloc(&d_a, MEMORY_SIZE);
    hipMalloc(&d_b, MEMORY_SIZE);
    hipMalloc(&d_c, MEMORY_SIZE);

    timer.onTimer(TIMER_TOTAL);

    timer.onTimer(TIMER_DATA_TRANSFER_H2D);
    hipMemcpy(d_a, a, MEMORY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, MEMORY_SIZE, hipMemcpyHostToDevice);
    timer.offTimer(TIMER_DATA_TRANSFER_H2D);

    timer.onTimer(TIMER_KERNEL);
    vectorAdd <<<dimGrid, dimBlock>>>(d_a, d_b, d_c, FLATTEN_MATRIX_SIZE);
    hipDeviceSynchronize();
    timer.offTimer(TIMER_KERNEL);

    timer.onTimer(TIMER_DATA_TRANSFER_D2H);
    hipMemcpy(c, d_c, MEMORY_SIZE, hipMemcpyDeviceToHost);
    timer.offTimer(TIMER_DATA_TRANSFER_D2H);

    timer.offTimer(TIMER_TOTAL);

    timer.onTimer(TIMER_HOST);
    for (int i = 0; i < FLATTEN_MATRIX_SIZE; i++) {
        h_c[i] = a[i] + b[i];
    }
    timer.offTimer(TIMER_HOST);

    bool isValidData = true;
    for (int i = 0; i < FLATTEN_MATRIX_SIZE; i++) {
        if (h_c[i] != c[i]) {
            printf("\t[%d] The result is not matched! (%d, %d)\n", i, h_c[i], c[i]);
            isValidData = false;
        }
    }

    if (isValidData) {
        printf("\tGPU works well!\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return isValidData;
}

int main(int argc, char **argv) {
    initData();
    initTimer();

    int count = 0;
    for (int gridX = 0; gridX <= LIMIT - 2; gridX++) {
        for (int gridY = 0; gridY <= LIMIT - gridX - 1; gridY++) {
            for (int blockX = 0; blockX <= LIMIT - gridX - gridY; blockX++) {
                int blockY = LIMIT - gridX - gridY - blockX;
                if (blockX + blockY <= 10 && gridY < 16) {
                    // blockX + blockY <= 1024 && gridY < 65536
                    dim3 dimBlock(1 << blockX, 1 << blockY, 1);
                    dim3 dimGrid(1 << gridX, 1 << gridY, 1);
                    printf("[#%d] ", ++count);
                    runVectorAdd(dimGrid, dimBlock);
                    // 가능한 모든 경우의 수에 대해서 테스트
                    timer.printToFile((char *) toReportFileName(gridX, gridY, blockX, blockY).c_str());
                }
            }
        }
    }

    freeData();
    return 0;
}
